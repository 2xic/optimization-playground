#define IS_CUDA 1 

#include <stdlib.h>
#include "stdio.h"
#include "kernel.h"


int main(){
    printf("Creating first matrix :)\n");

	Matrix *d_a;
    createMatrix<<<1, 1>>>((void**)&d_a, 2, 2);
    setElement<<<1, 1>>>(d_a, 0, 0, -2);
    setElement<<<1, 1>>>(d_a, 0, 1, 1);
    setElement<<<1, 1>>>(d_a, 1, 0, 0);
    setElement<<<1, 1>>>(d_a, 1, 1, 4);

    printf("Creating second matrix :)\n");

	Matrix *d_b;
    createMatrix<<<1, 1>>>((void**)&d_b, 2, 2);
    setElement<<<1, 1>>>(d_b, 0, 0, -2);
    setElement<<<1, 1>>>(d_b, 0, 1, 1);
    setElement<<<1, 1>>>(d_b, 1, 0, 0);
    setElement<<<1, 1>>>(d_b, 1, 1, 4);

    printf("Creating last matrix :)\n");

    Matrix *d_c;
    createMatrix<<<2, 1>>>((void**)&d_c, 2, 2);
    hipMemset(d_c->data, 0, 2 * 2);

    MatMul<<<2, 1>>>(d_a, d_b, d_c);
    printf("Finished  matmul :D\n");


    int *out;
    printf("copy ?? ");
    out = (int*) malloc(sizeof(int *) * 2 * 2);
    hipMemcpy(out, d_c->data, sizeof(int) * 2 * 2, hipMemcpyDeviceToHost);
    printf("printing ?? ");
    print_array(out, 2);

    printf("done \n");

    hipFree(d_a->data);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_b->data);
    hipFree(d_c);
    hipFree(d_c->data);

///    MatMul<<<1, 1>>>(d_a, d_b, d_c);

    /*
    Matrix *results;
    createMatrix<<<1,1>>>(
        a->rows,
        b->columns
    );
    */


    /*
    float *a, *b, *out; 
    float *d_a, *d_b, *d_out;
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    for(int i = 0; i < N; i++){
        a[i] = 1.0f; 
        b[i] = 2.0f;
    }

    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    out = (float*)malloc(sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);
   // hipMemcpy(d_out, out, sizeof(float) * N, hipMemcpyHostToDevice);

    vector_add<<<1,1>>>(d_out, d_a, d_b, N);

    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);
    
    free(a);
    hipFree(d_a);
    free(b);
    hipFree(d_b);    
    free(out);
    hipFree(d_out);
    */
}
