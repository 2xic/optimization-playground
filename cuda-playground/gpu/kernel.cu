#include "hip/hip_runtime.h"
#include <stdlib.h>
#include "kernel.h"
#include <stdio.h>

__device__ float a_item;
__device__ float b_item;

void print_array(float *ptr, int length)
{
    printf("[");
    for (int i = 0; i < length; i++)
    {
        if (i > 0)
        {
            printf(", ");
        }
        printf("%f", ptr[i]);
    }
    printf("]");
}

Matrix *createMatrixGpu(int rows, int columns)
{
    float *a_device;
    int SIZE = rows * columns;
    Matrix *a = (Matrix *)malloc(sizeof(Matrix));
    hipMalloc(&a_device, SIZE * sizeof(float));
    a->data = a_device;
    a->rows = rows;
    a->columns = columns;
    a->device = 1;

    return a;
}

extern "C" void sendToHost(Matrix *m)
{
    // printf("Sending it to host \n");
    float *c_host;
    int SIZE = m->rows * m->columns;
    c_host = (float *)malloc(SIZE * sizeof(float));

    hipMemcpy(c_host, m->data, SIZE * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(m->data);

    m->data = c_host;
    m->device = 0;
}

extern "C" void sendToGpu(Matrix *m)
{
    //  printf("Sending it to host \n");
    int SIZE = m->rows * m->columns;
    float *c_device;
    hipMalloc(&c_device, SIZE * sizeof(float));

    hipMemcpy(c_device, m->data, SIZE * sizeof(float), hipMemcpyHostToDevice);
    free(m->data);

    m->data = c_device;
    m->device = 1;
}

extern "C" Matrix *GpuMatrixMatMul(Matrix *a, Matrix *b)
{
    Matrix *c = createMatrixGpu(a->rows, b->columns);
    MatMul<<<1, 1>>>(a->data, b->data, c->data, a->rows, b->columns);

    return c;
}

__global__ void setElement(float *data, int columns, int row, int col, float value)
{
    int rowIndex = columns * row;
    data[rowIndex + col] = value;
}

__device__ void getElement(float *data, int row, int colsize, int col, float *value)
{
    int row_idx = row * colsize;
    *value = data[row_idx + col];
}

__global__ void SimpleMatrixOperator(float *a, float *b, float constant, float *c, int rows, int cols, int operator_val)
{
    auto get = [](int cols, int i, int j, float *M, float C, float *res)
    {
        if (M != NULL)
        {
            getElement(M, cols, i, j, res);
        }
        else
        {
            *res = C;
        }
    };

    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            float value = 0;

            if (operator_val == ADD)
            {
                get(cols, i, j, a, constant, &a_item);
                get(cols, i, j, b, constant, &b_item);
                value = a_item + b_item;
            }
            else if (operator_val == SUB)
            {
                get(cols, i, j, a, constant, &a_item);
                get(cols, i, j, b, constant, &b_item);
                value = a_item - b_item;
            }
            else if (operator_val == MUL)
            {
                get(cols, i, j, a, constant, &a_item);
                get(cols, i, j, b, constant, &b_item);
                value = a_item * b_item;
            }
            else if (operator_val == DIV)
            {
                get(cols, i, j, a, constant, &a_item);
                get(cols, i, j, b, constant, &b_item);
                value = a_item / b_item;
            }

            setElement<<<1, 1>>>(
                c,
                cols,
                i,
                j,
                value);
        }
    }
}

__global__ void FastSimpleMatrixAddOperator(float *a, float *b, float constant, float *c, int rows, int cols, int operator_val)
{
    auto get = [](int cols, int i, int j, float *M, float C, float *res)
    {
        if (M != NULL)
        {
            getElement(M, cols, i, j, res);
        }
        else
        {
            *res = C;
        }
    };
    auto set = [](float *data, int columns, int row, int col, float value)
    {
        int rowIndex = columns * row;
        data[rowIndex + col] = value;
    };

    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    float value = 0;

    if (operator_val == ADD)
    {
        get(cols, i, j, a, constant, &a_item);
        get(cols, i, j, b, constant, &b_item);
        value = a_item + b_item;
    }

    set(
        c,
        cols,
        i,
        j,
        value);
}

extern "C" Matrix *GpuAdd(Matrix *a, Matrix *b)
{
    Matrix *c = createMatrixGpu(a->rows, b->columns);

    dim3 dimBlock(a->rows, b->columns);
    dim3 dimGrid(1, 1);

    FastSimpleMatrixAddOperator<<<dimGrid, dimBlock>>>(a->data, b->data, -1, c->data, a->rows, b->columns, ADD);
    //    SimpleMatrixOperator<<<1, 1>>>(a->data, b->data, -1, c->data, a->rows, b->columns, ADD);

    return c;
}

extern "C" Matrix *GpuAddConstant(Matrix *a, float b, int direction)
{
    Matrix *c = createMatrixGpu(a->rows, a->columns);
    SimpleMatrixOperator<<<1, 1>>>(a->data, nullptr, b, c->data, a->rows, a->columns, ADD);

    return c;
}

extern "C" Matrix *GpuMul(Matrix *a, Matrix *b)
{
    Matrix *c = createMatrixGpu(a->rows, b->columns);
    SimpleMatrixOperator<<<1, 1>>>(a->data, b->data, -1, c->data, a->rows, b->columns, MUL);

    return c;
}

extern "C" Matrix *GpuMulConstant(Matrix *a, float b, int direction)
{
    Matrix *c = createMatrixGpu(a->rows, a->columns);
    SimpleMatrixOperator<<<1, 1>>>(a->data, nullptr, b, c->data, a->rows, a->columns, MUL);

    return c;
}

extern "C" Matrix *GpuDivideConstant(Matrix *a, float b, int direction)
{
    Matrix *c = createMatrixGpu(a->rows, a->columns);
    SimpleMatrixOperator<<<1, 1>>>(a->data, nullptr, b, c->data, a->rows, a->columns, DIV);

    return c;
}

extern "C" Matrix *GpuSubtract(Matrix *a, Matrix *b)
{
    Matrix *c = createMatrixGpu(a->rows, b->columns);
    SimpleMatrixOperator<<<1, 1>>>(a->data, b->data, -1, c->data, a->rows, b->columns, SUB);

    return c;
}

extern "C" Matrix *GpuSubtractConstant(Matrix *a, float b, int direction)
{
    Matrix *c = createMatrixGpu(a->rows, a->columns);
    SimpleMatrixOperator<<<1, 1>>>(a->data, nullptr, b, c->data, a->rows, a->columns, SUB);

    return c;
}

__global__ void _transpose(float *target, float *source, int columns, int rows)
{
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < columns; j++)
        {
            getElement(source, columns, i, j, &a_item);
            //   printf("Hei! %f\n", a_item);
            setElement<<<1, 1>>>(target, columns, j, i, a_item);
        }
    }
}

extern "C" Matrix *GpuTranspose(Matrix *a)
{
    // printf("Test! (%i, %i)\n", a->columns, a->rows);

    Matrix *c = createMatrixGpu(a->columns, a->rows);
    _transpose<<<1, 1>>>(c->data, a->data, a->columns, a->rows);

    return c;
}

__global__ void _Exp(float *target, float *source, int columns, int rows)
{
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < columns; j++)
        {
            getElement(source, columns, i, j, &a_item);
            //   printf("Hei! %f\n", a_item);
            setElement<<<1, 1>>>(target, columns, j, i, exp(a_item));
        }
    }
}

extern "C" Matrix *GpuExp(Matrix *a)
{
    // printf("Test! (%i, %i)\n", a->columns, a->rows);

    Matrix *c = createMatrixGpu(a->columns, a->rows);
    _Exp<<<1, 1>>>(c->data, a->data, a->columns, a->rows);

    return c;
}

// Add the remaning operators = Victory :)

// https://developer.nvidia.com/blog/cuda-dynamic-parallelism-api-principles/
// https://stackoverflow.com/questions/49687130/pass-by-reference-in-device-function-cuda
//  -> Looks like there is some improved synchronization I can do
__device__ float accumulator;
__global__ void MatMul(float *a, float *b, float *c, int columns, int rows)
{
    //    printf("hello world \n");
    for (int row = 0; row < rows; row++)
    {
        for (int column = 0; column < columns; column++)
        {
            accumulator = 0;
            // float accumulator = 0;
            for (int current_colum = 0; current_colum < columns; current_colum++)
            {
                getElement(a, columns, row, current_colum, &a_item);
                getElement(b, columns, current_colum, column, &b_item);

                accumulator += a_item * b_item;
            }
            setElement<<<1, 1>>>(
                c,
                columns,
                row,
                column,
                accumulator);
        }
    }
}
