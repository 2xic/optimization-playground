#include "hip/hip_runtime.h"
#include <stdlib.h>
#include "kernel.h"
#include <stdio.h>


__global__ void MatMul(float *a, float *b, float *c, int columns, int rows);

void print_array(float *ptr, int length)
{
    printf("[");
    for (int i = 0; i < length; i++)
    {
        if (i > 0)
        {
            printf(", ");
        }
        printf("%f", ptr[i]);
    }
    printf("]");
}

Matrix *createMatrixGpu(int rows, int columns) {
    // -> 
    printf("Sending it to GPU\n");
    float *a_device;
    int SIZE = rows * columns;
    Matrix *a = (Matrix*)malloc(sizeof(Matrix));
    hipMalloc(&a_device, SIZE * sizeof(float));
    a->data = a_device;
    a->rows = rows;
    a->columns = columns;
    a->device = 1;

    return a;
}

extern "C" void sendToHost(Matrix *m) {
    printf("Sending it to host \n");
    float *c_host;
    int SIZE = m->rows * m->columns;
    c_host = (float*)malloc(SIZE * sizeof(float));

    hipMemcpy(c_host, m->data, SIZE * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(m->data);

    m->data = c_host;
    m->device = 0;
}

extern "C" void sendToGpu(Matrix *m) {
    printf("Sending it to host \n");
    int SIZE = m->rows * m->columns;
    float *c_device;
    hipMalloc(&c_device, SIZE * sizeof(float));

    hipMemcpy(c_device, m->data, SIZE * sizeof(float), hipMemcpyHostToDevice);
    free(m->data);

    m->data = c_device;
    m->device = 1;
}

extern "C" Matrix* MatrixMatMul(Matrix*a, Matrix*b) {
    // Results
    Matrix *c = createMatrixGpu(a->rows, b->columns);
    MatMul<<<1, 1>>>(a->data, b->data, c->data, a->rows, b->columns);

    return c;
}

__global__ void setElement(float *data, int columns, int row, int col, float value){
    int rowIndex = columns * row;
    data[rowIndex + col] = value;
}


__global__ void MatMul(float *a, float *b, float *c, int columns, int rows)
{
    printf("hello world \n");
    for (int row = 0; row < rows; row++)
    {
        for (int column = 0; column < columns; column++)
        {
            float accumulator = 0;
            for (int current_colum = 0; current_colum < columns; current_colum++)
            {
                // current row + current column
                int a_rowIndex = columns * row;
                float a_item = a[a_rowIndex + current_colum];

                // current column + column
                int b_rowIndex = columns * current_colum;
                float b_item = b[b_rowIndex + column];

                accumulator += a_item * b_item;
            }
            setElement<<<1,1>>>(
                c,
                columns,
                row,
                column,
                accumulator
            );
        }
    }
}
