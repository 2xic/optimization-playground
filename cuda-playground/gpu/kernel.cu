#include "hip/hip_runtime.h"
#include <stdlib.h>
#include "kernel.h"
#include <stdio.h>

void print_array(int *ptr, int length)
{
    printf("[");
    for (int i = 0; i < length; i++)
    {
        if (i > 0)
        {
            printf(", ");
        }
        printf("%i", ptr[i]);
    }
    printf("]");
}

__global__ void setElement(int *data, int columns, int row, int col, int value){
    int rowIndex = columns * row;
    data[rowIndex + col] = value;
}

__global__ void MatMul(int *a, int *b, int *c, int columns, int rows)
{
    printf("hello world \n");
    for (int row = 0; row < rows; row++)
    {
        for (int column = 0; column < columns; column++)
        {
            int accumulator = 0;
            for (int current_colum = 0; current_colum < columns; current_colum++)
            {
                // current row + current column
                int a_rowIndex = columns * row;
                int a_item = a[a_rowIndex + current_colum];

                // current column + column
                int b_rowIndex = columns * current_colum;
                int b_item = b[b_rowIndex + column];

                accumulator += a_item * b_item;
            }
            setElement<<<1,1>>>(
                c,
                columns,
                row,
                column,
                accumulator
            );
        }
    }
}
