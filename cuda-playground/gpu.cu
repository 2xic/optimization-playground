#include "hip/hip_runtime.h"
#include <stdlib.h>
#include "stdio.h"
#include "shared.c"

#define N 1000

__global__ void vector_add(float *out, float *a, float *b, int n) {
    for(int i = 0; i < N; i++){
        out[i] = a[i] + b[i];
    }
}

int main(){
    float *a, *b, *out; 
    float *d_a, *d_b, *d_out;
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    for(int i = 0; i < N; i++){
        a[i] = 1.0f; 
        b[i] = 2.0f;
    }

    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    out = (float*)malloc(sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);
   // hipMemcpy(d_out, out, sizeof(float) * N, hipMemcpyHostToDevice);

    vector_add<<<1,1>>>(d_out, d_a, d_b, N);

    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    printf("Hello World from GPU!\n");
    print_array(out, N);
    
    free(a);
    hipFree(d_a);
    free(b);
    hipFree(d_b);    
    free(out);
    hipFree(d_out);
}
